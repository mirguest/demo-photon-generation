#include "hip/hip_runtime.h"

#include "scintillation.hh"
#include "GPUOpEngine.hh"
#include "GPUParametersManager.hh"

typedef scintillation<DummyGpuOpEngine, DummyGpuParametersManager> DummyGpuScintillation_t;

DummyGpuParametersManager_t parameters_manager;
DummyGpuScintillation_t scint(parameters_manager);

__global__
void launch_kernel() {
    scint.generate();
}

int main() {
    const int N = 1024;
    launch_kernel<<<1, N>>>();
}
